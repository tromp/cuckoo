#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include <chrono>

#ifdef _WIN32
#include "../windows/getopt.h"
#else
#include <unistd.h>
#endif

#include "cuckoo.h"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"

typedef uint8_t u8;
typedef uint16_t u16;

typedef u32 node_t;
typedef u64 nonce_t;

typedef std::chrono::milliseconds ms;

#ifndef XBITS
#define XBITS ((EDGEBITS-16)/2)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES ((node_t)1 << NODEBITS)
#define NODEMASK (NNODES - 1)

const static u32 NX        = 1 << XBITS;
const static u32 NX2       = NX * NX;
const static u32 XMASK     = NX - 1;
const static u32 X2MASK    = NX2 - 1;
const static u32 YBITS     = XBITS;
const static u32 NY        = 1 << YBITS;
const static u32 YZBITS    = EDGEBITS - XBITS;
const static u32 NYZ       = 1 << YZBITS;
const static u32 ZBITS     = YZBITS - YBITS;
const static u32 NZ        = 1 << ZBITS;

#define EPS_A 133/128
#define EPS_B 85/128

const static u32 ROW_EDGES_A = NYZ * EPS_A;
const static u32 ROW_EDGES_B = NYZ * EPS_B;

const static u32 EDGES_A = ROW_EDGES_A / NX;
const static u32 EDGES_B = ROW_EDGES_B / NX;

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ __forceinline__ ulonglong4 Pack8(const u32 e0, const u32 e1, const u32 e2, const u32 e3, const u32 e4, const u32 e5, const u32 e6, const u32 e7) {
  return make_ulonglong4((u64)e0<<32|e1, (u64)e2<<32|e3, (u64)e4<<32|e5, (u64)e6<<32|e7);
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, int * __restrict__ indexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;
  const int FLUSHA2 = 2*FLUSHA;

  __shared__ EdgeOut tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  for (int row = lid; row < NX; row += dim)
    counters[row] = 0;
  __syncthreads();

  const int col = group % NX;
  const int loops = NEDGES / nthreads;
  for (int i = 0; i < loops; i++) {
    u32 nonce = gid * loops + i;
    u32 node1, node0 = dipnode(sipkeys, (u64)nonce, 0);
    if (sizeof(EdgeOut) == sizeof(uint2))
      node1 = dipnode(sipkeys, (u64)nonce, 1);
    int row = node0 & XMASK;
    int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1));
    tmp[row][counter] = make_Edge(nonce, tmp[0][0], node0, node1);
    __syncthreads();
    if (counter == FLUSHA-1) {
      int localIdx = min(FLUSHA2, counters[row]);
      int newCount = localIdx % FLUSHA;
      int nflush = localIdx - newCount;
      int cnt = min((int)atomicAdd(indexes + row * NX + col, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        buffer[((u64)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[row][t] = tmp[row][t + nflush];
      }
      counters[row] = newCount;
    }
    __syncthreads();
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      buffer[((u64)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(u32 nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut, typename EdgeOut>
__global__ void SeedB(const siphash_keys &sipkeys, const EdgeOut * __restrict__ source, ulonglong4 * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ EdgeOut tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
  __shared__ int counters[NX];

  // if (group>=0&&lid==0) print_log("group  %d  -\n", group);
  for (int col = lid; col < NX; col += dim)
    counters[col] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)sourceIndexes[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * maxOut + edgeIndex;
      EdgeOut edge = __ldg(&source[index]);
      if (null(edge)) continue;
      u32 node1 = endpoint(sipkeys, edge, 0);
      col = (node1 >> XBITS) & XMASK;
      counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1));
      tmp[col][counter] = edge;
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        destination[((u64)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + nflush];
      }
      counters[col] = newCount;
    }
    __syncthreads();
  }
  EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      destination[((u64)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void Increase2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  u32 old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + NZ/32, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  return (ecounters[word + NZ/32] & mask) != 0;
}

__device__ uint2 make_Edge(const u32 nonce, const uint2 dummy, const u32 node0, const u32 node1) {
   return make_uint2(node0, node1);
}

__device__ uint2 make_Edge(const uint2 edge, const uint2 dummy, const u32 node0, const u32 node1) {
   return edge;
}

__device__ u32 make_Edge(const u32 nonce, const u32 dummy, const u32 node0, const u32 node1) {
   return nonce;
}

template <typename Edge> u32 __device__ endpoint(const siphash_keys &sipkeys, Edge e, int uorv);

__device__ u32 endpoint(const siphash_keys &sipkeys, u32 nonce, int uorv) {
  return dipnode(sipkeys, nonce, uorv);
}

__device__ u32 endpoint(const siphash_keys &sipkeys, uint2 nodes, int uorv) {
  return uorv ? nodes.y : nodes.x;
}

template<int maxIn, typename EdgeIn, int maxOut, typename EdgeOut>
__global__ void Round(const int round, const siphash_keys &sipkeys, const EdgeIn * __restrict__ source, EdgeOut * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const static int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

  __shared__ u32 ecounters[COUNTERWORDS];

  for (int i = lid; i < COUNTERWORDS; i += dim)
    ecounters[i] = 0;
  __syncthreads();
  const int edgesInBucket = min(sourceIndexes[group], maxIn);
  const int loops = (edgesInBucket + dim-1) / dim;

  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&source[index]);
      if (null(edge)) continue;
      u32 node = endpoint(sipkeys, edge, round&1);
      Increase2bCounter(ecounters, node >> (2*XBITS));
    }
  }
  __syncthreads();
  for (int loop = 0; loop < loops; loop++) {
    const int lindex = loop * dim + lid;
    if (lindex < edgesInBucket) {
      const int index = maxIn * group + lindex;
      EdgeIn edge = __ldg(&source[index]);
      if (null(edge)) continue;
      u32 node0 = endpoint(sipkeys, edge, round&1);
      if (Read2bCounter(ecounters, node0 >> (2*XBITS))) {
        u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
        const int bucket = node1 & X2MASK;
        const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), maxOut - 1);
        destination[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *destination, node1, node0)
                                                          : make_Edge(edge, *destination, node0, node1);
      }
    }
  }
  // if (group==0&&lid==0) print_log("round %d cnt(0,0) %d\n", round, sourceIndexes[0]);
}

template<int maxIn>
__global__ void Tail(const uint2 *source, uint2 *destination, const int *sourceIndexes, int *destinationIndexes) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  int myEdges = sourceIndexes[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(destinationIndexes, myEdges);
  __syncthreads();
  for (int i = lid; i < myEdges; i += dim)
    destination[destIdx + lid] = source[group * maxIn + lid];
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d\0", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];

  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < loops; i++) {
    u64 nonce = gid * loops + i;
    u64 u = dipnode(sipkeys, nonce, 0);
    u64 v = dipnode(sipkeys, nonce, 1);
    for (int i = 0; i < PROOFSIZE; i++) {
      if (recoveredges[i].x == u && recoveredges[i].y == v)
        nonces[i] = nonce;
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

struct trimparams {
  u16 expand;
  u16 ntrims;
  blockstpb genA;
  blockstpb genB;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    expand              =    0;
    ntrims              =  176;
    genA.blocks         = 4096;
    genA.tpb            =  256;
    genB.blocks         =  NX2;
    genB.tpb            =  128;
    trim.blocks         =  NX2;
    trim.tpb            =  512;
    tail.blocks         =  NX2;
    tail.tpb            = 1024;
    recover.blocks      = 1024;
    recover.tpb         = 1024;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX * NY * sizeof(u32);
  ulonglong4 *bufferA;
  ulonglong4 *bufferB;
  ulonglong4 *bufferAB;
  int *indexesE;
  int *indexesE2;
  u32 hostA[NX * NY];
  u32 *uvnodes;
  siphash_keys sipkeys, *dipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    tp = _tp;
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
    checkCudaErrors_V(hipMalloc((void**)&indexesE, indexesSize));
    checkCudaErrors_V(hipMalloc((void**)&indexesE2, indexesSize));
    sizeA = ROW_EDGES_A * NX * (tp.expand > 0 ? sizeof(u32) : sizeof(uint2));
    sizeB = ROW_EDGES_B * NX * (tp.expand > 1 ? sizeof(u32) : sizeof(uint2));
    const size_t bufferSize = sizeA + sizeB;
    checkCudaErrors_V(hipMalloc((void**)&bufferA, bufferSize));
    bufferB  = bufferA + sizeA / sizeof(ulonglong4);
    bufferAB = bufferA + sizeB / sizeof(ulonglong4);
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  u64 globalbytes() const {
    return (sizeA+sizeB) + 2 * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferA));
    checkCudaErrors_V(hipFree(indexesE2));
    checkCudaErrors_V(hipFree(indexesE));
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  u32 trim() {
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEvent_t startall, stopall;
    checkCudaErrors(hipEventCreate(&startall)); checkCudaErrors(hipEventCreate(&stopall));
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));

    hipMemset(indexesE, 0, indexesSize);
    hipMemset(indexesE2, 0, indexesSize);
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);

    if (tp.expand == 0)
      SeedA<EDGES_A, uint2><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);
    else
      SeedA<EDGES_A,   u32><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    if (abort) return false;
    hipEventRecord(start, NULL);

    const u32 halfA = sizeA/2 / sizeof(ulonglong4);
    const u32 halfE = NX2 / 2;
    if (tp.expand == 0) {
      SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
      SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
    } else {
      SeedB<EDGES_A,   u32><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const   u32 *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
      SeedB<EDGES_A,   u32><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const   u32 *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
    }

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    print_log("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);
    if (abort) return false;

    hipMemset(indexesE, 0, indexesSize);

    if (tp.expand == 0)
      Round<EDGES_A, uint2, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    else if (tp.expand == 1)
      Round<EDGES_A,   u32, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    else // tp.expand == 2
      Round<EDGES_A,   u32, EDGES_B,   u32><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)bufferA, (  u32 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
    if (abort) return false;

    hipMemset(indexesE2, 0, indexesSize);

    if (tp.expand < 2)
      Round<EDGES_B, uint2, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296
    else
      Round<EDGES_B,   u32, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const   u32 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296

    if (abort) return false;
    hipMemset(indexesE, 0, indexesSize);
    Round<EDGES_B/2, uint2, EDGES_A/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(2, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .176
    if (abort) return false;
    hipMemset(indexesE2, 0, indexesSize);
    Round<EDGES_A/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(3, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .117

    hipDeviceSynchronize();

    for (int round = 4; round < tp.ntrims; round += 2) {
      if (abort) return false;
      hipMemset(indexesE, 0, indexesSize);
      Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round, *dipkeys,  (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
      if (abort) return false;
      hipMemset(indexesE2, 0, indexesSize);
      Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round+1, *dipkeys,  (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2);
    }

    if (abort) return false;
    hipMemset(indexesE, 0, indexesSize);
    hipDeviceSynchronize();

    Tail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
    hipMemcpy(hostA, indexesE, NX * NY * sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return hostA[0];
  }
};

#define IDXSHIFT 10
#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = new u64[CUCKOO_SIZE];
  }
  ~cuckoo_hash() {
    delete[] cuckoo;
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

const static u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

const static u32 MAXEDGES = 0x20000;

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  cuckoo_hash *cuckoo;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices
  u32 us[MAXPATHLEN];
  u32 vs[MAXPATHLEN];

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp) {
    edges   = new uint2[MAXEDGES];
    cuckoo  = new cuckoo_hash();
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce) {
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    }
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete cuckoo;
    delete[] edges;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    soledges[i].x = u2/2;
    soledges[i].y = v2/2;
  }

  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
    recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    assert(ni == PROOFSIZE);
    sols.resize(sols.size() + PROOFSIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(soledges));
    hipMemset(trimmer.indexesE2, 0, trimmer.indexesSize);
    Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, trimmer.bufferA, (int *)trimmer.indexesE2);
    hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesE2, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
    checkCudaErrors_V(hipDeviceSynchronize());
    qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  u32 path(u32 u, u32 *us) {
    u32 nu, u0 = u;
    for (nu = 0; u; u = (*cuckoo)[u]) {
      if (nu >= MAXPATHLEN) {
        while (nu-- && us[nu] != u) ;
        if (~nu) {
          print_log("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
          exit(0);
        }
        print_log("maximum path length exceeded\n");
        return 0; // happens once in a million runs or so; signal trouble
      }
      us[nu++] = u;
    }
    return nu;
  }

  void addedge(uint2 edge) {
    const u32 u0 = edge.x << 1, v0 = (edge.y << 1) | 1;
    if (u0) {
      u32 nu = path(u0, us), nv = path(v0, vs);
      if (!nu-- || !nv--)
        return; // drop edge causing trouble
      // print_log("vx %02x ux %02x e %08x uxyz %06x vxyz %06x u0 %x v0 %x nu %d nv %d\n", vx, ux, e, uxyz, vxyz, u0, v0, nu, nv);
      if (us[nu] == vs[nv]) {
        const u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        const u32 len = nu + nv + 1;
        print_log("%4d-cycle found\n", len);
        if (len == PROOFSIZE)
          solution(us, nu, vs, nv);
        // if (len == 2) print_log("edge %x %x\n", edge.x, edge.y);
      } else if (nu < nv) {
        while (nu--)
          cuckoo->set(us[nu+1], us[nu]);
        cuckoo->set(u0, v0);
      } else {
        while (nv--)
          cuckoo->set(vs[nv+1], vs[nv]);
        cuckoo->set(v0, u0);
      }
    }
  }

  void findcycles(uint2 *edges, u32 nedges) {
    memset(cuckoo->cuckoo, 0, CUCKOO_SIZE * sizeof(u64));
    for (u32 i = 0; i < nedges; i++)
      addedge(edges[i]);
  }

  int solve() {
    u32 timems,timems2;
    auto time0 = std::chrono::high_resolution_clock::now();

    trimmer.abort = false;
    u32 nedges = trimmer.trim();
    if (!nedges)
      return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, nedges * 8, hipMemcpyDeviceToHost);
    auto time1 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<ms>(time1 - time0);
    timems = duration.count();
    time0 = std::chrono::high_resolution_clock::now();
    findcycles(edges, nedges);
    time1 = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<ms>(time1 - time0);
    timems2 = duration.count();
    print_log("findcycles edges %d time %d ms total %d ms\n", nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }
};

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++)
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, &ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.expand = params->expand;
  tp.genA.blocks = params->genablocks;
  tp.genA.tpb = params->genatpb;
  tp.genB.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
  assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  hipSetDevice(params->device);

  SolverCtx* ctx = new SolverCtx(tp, params->mutate_nonce);

  return ctx;
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->expand = tp.expand;
  params->genablocks = tp.genA.blocks;
  params->genatpb = tp.genA.tpb;
  params->genbtpb = tp.genB.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = tp.recover.blocks;
  params->recovertpb = tp.recover.tpb;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "sb:c:d:E:h:k:m:n:r:U:u:v:w:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-d device] [-E 0-2] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedAblocks] [-u seedAthreads] [-v seedBthreads] [-w Trimthreads] [-y Tailthreads] [-Z recoverblocks] [-z recoverthreads]\n", NODEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -E %d -h \"\" -m %d -n %d -r %d -U %d -u %d -v %d -w %d -y %d -Z %d -z %d\n", NODEBITS, device, tp.expand, tp.ntrims, nonce, range, tp.genA.blocks, tp.genA.tpb, tp.genB.tpb, tp.trim.tpb, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'd':
        params.device = atoi(optarg);
        break;
      case 'E':
        params.expand = atoi(optarg);
        assert(params.expand <= 2);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        params.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        params.genablocks = atoi(optarg);
        break;
      case 'u':
        params.genatpb = atoi(optarg);
        break;
      case 'v':
        params.genbtpb = atoi(optarg);
        break;
      case 'w':
        params.trimtpb = atoi(optarg);
        break;
      case 'y':
        params.tailtpb = atoi(optarg);
        break;
      case 'Z':
        params.recoverblocks = atoi(optarg);
        break;
      case 'z':
        params.recovertpb = atoi(optarg);
        break;
    }
  }
  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);

  print_log("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 10240; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
