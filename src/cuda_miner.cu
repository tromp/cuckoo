#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2015 John Tromp

// The edge=trimming time-memory trade-off is due to Dave Anderson:
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html



#include <stdint.h>
#include <string.h>
#include <time.h>
#include "cuckoo.h"
#include "hip/hip_runtime.h"
#include ""

#if SIZESHIFT <= 32
typedef u32 nonce_t;
typedef u32 node_t;
#else
typedef u64 nonce_t;
typedef u64 node_t;
#endif
#include <openssl/sha.h>

__shared__ siphash_ctx shared_siphash_ctx;

// d(evice s)ipnode
__device__ node_t dipnode(nonce_t nce, u32 uorv) {
#if (__CUDA_ARCH__  < 320)
	u64 nonce = 2*nce + uorv;
	u64 v0 = shared_siphash_ctx.v[0], v1 = shared_siphash_ctx.v[1], v2 = shared_siphash_ctx.v[2], v3 = shared_siphash_ctx.v[3] ^ nonce;

	SIPROUND; SIPROUND;
	v0 ^= nonce;
	v2 ^= 0xff;
	SIPROUND; SIPROUND; SIPROUND; SIPROUND;
	return (v0 ^ v1 ^ v2  ^ v3) & NODEMASK;
#else
	uint2 nonce = vectorize((nce << 1) + uorv);
	uint2 v0, v1, v2, v3;
	v0 = shared_siphash_ctx.vv[0], v1 = shared_siphash_ctx.vv[1], v2 = shared_siphash_ctx.vv[2], v3 = shared_siphash_ctx.vv[3] ^ nonce;

	SIPROUND2; SIPROUND2;
	v0 ^= nonce;
	v2 ^= vectorize(0xff);
	SIPROUND2; SIPROUND2; SIPROUND2; SIPROUND2;
	return devectorize(v0 ^ v1 ^ v2  ^ v3) & NODEMASK;
#endif
}

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (SIZE >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// SIZE >> IDXSHIFT == HALFSIZE >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif
// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (SIZESHIFT/3))

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(nonce_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(node_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (HALFSIZE >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(node_t u) {
	  node_t idx = u / 16;
	  u32 bit = 1 << (2 * (u % 16));
	  //u32 old = atomicOr(&bits[idx], bit); // ~1% slower than 2 lines below
	  u32 old = bits[idx];
	  bits[idx] |= bit;
	  //
	  u32 bit2 = bit << 1;
	  if ((old & (bit2 | bit)) == bit) atomicOr(&bits[idx], bit2); //somehow this is faster
  } 
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (SIZE >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by SIZESHIFT
#define KEYBITS (64-SIZESHIFT)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
    assert(cuckoo != 0);
  }
  ~cuckoo_hash() {
    free(cuckoo);
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << SIZESHIFT | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> SIZESHIFT) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> SIZESHIFT) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> SIZESHIFT) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & (SIZE-1));
      }
    }
  }
};

class cuckoo_ctx {
public:
  siphash_ctx sip_ctx;
  shrinkingset alive;
  twice_set nonleaf;
  int nthreads;

  cuckoo_ctx(const char* header, u32 n_threads) {
    setheader(&sip_ctx, header);
    nthreads = n_threads;
  }
};

#define TPB 128

__global__ void
__launch_bounds__(TPB, 1)
count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  
  if (threadIdx.x == 0) {
	  shared_siphash_ctx.v[0] = ctx->sip_ctx.v[0];
	  shared_siphash_ctx.v[1] = ctx->sip_ctx.v[1];
	  shared_siphash_ctx.v[2] = ctx->sip_ctx.v[2];
	  shared_siphash_ctx.v[3] = ctx->sip_ctx.v[3];
  }
  __syncthreads();
  
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (nonce_t block = id*32; block < HALFSIZE; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);

    for (nonce_t nonce = block; alive32; alive32>>=1, nonce++) {
      if (alive32 & 1) {
        node_t u = dipnode(nonce, uorv);
        if ((u & PART_MASK) == part) {
          nonleaf.set(u >> PART_BITS);
        }
      }
    }
  }
}

__global__ void
__launch_bounds__(TPB, 1)
kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  
  if (threadIdx.x == 0) {
	  shared_siphash_ctx.v[0] = ctx->sip_ctx.v[0];
	  shared_siphash_ctx.v[1] = ctx->sip_ctx.v[1];
	  shared_siphash_ctx.v[2] = ctx->sip_ctx.v[2];
	  shared_siphash_ctx.v[3] = ctx->sip_ctx.v[3];
  }
  __syncthreads();
  
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (nonce_t block = id*32; block < HALFSIZE; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (nonce_t nonce = block; alive32; alive32>>=1, nonce++) {
      if (alive32 & 1) {
        node_t u = dipnode(nonce, uorv);
        if ((u & PART_MASK) == part) {
          if (!nonleaf.test(u >> PART_BITS)) {
            alive.reset(nonce);
          }
        }
      }
    }
  }
}

u32 path(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (++nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      exit(0);
    }
    us[nu] = u;
  }
  return nu;
}

typedef std::pair<node_t,node_t> edge;

#ifndef WIN32
#include <unistd.h>
#else
#include "getopt/getopt.h"
#endif

int main(int argc, char **argv) {
  int nthreads = 1;
  int ntrims   = 1 + (PART_BITS+3)*(PART_BITS+4)/2;
  const char *header = "";
  bool profiling = false;
  int c;
  while ((c = getopt (argc, argv, "h:m:n:t:p")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        ntrims = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
	  case 'p':
		  profiling = true;
		  break;
    }
  }
  printf("Looking for %d-cycle on cuckoo%d(\"%s\") with 50%% edges, %d trims, %d threads\n",
               PROOFSIZE, SIZESHIFT, header, ntrims, nthreads);
  u64 edgeBytes = HALFSIZE/8, nodeBytes = TWICE_WORDS*sizeof(u32);

  cuckoo_ctx ctx(header, nthreads);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  printf("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));
  hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);

  hipEvent_t start, stop;

  if (profiling) {  
	  checkCudaErrors(hipEventCreate(&start));
	  checkCudaErrors(hipEventCreate(&stop));
	  hipEventRecord(start, nullptr);
  }

  for (u32 round=0; round < ntrims; round++) {
    for (u32 uorv = 0; uorv < 2; uorv++) {
      for (u32 part = 0; part <= PART_MASK; part++) {
        checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
		count_node_deg << <nthreads / TPB, TPB >> >(device_ctx, uorv, part);
		kill_leaf_edges << <nthreads / TPB, TPB >> >(device_ctx, uorv, part);
		hipDeviceSynchronize();
      }
    }
  }
  if (profiling) {
	  hipEventRecord(stop, nullptr);
	  hipEventSynchronize(stop);

	  float duration;
	  hipEventElapsedTime(&duration, start, stop);
	  printf("%d rounds completed in %.2f seconds.\n", ntrims, duration / 1000.0f);
  }
  u32 *bits;
  bits = (u32 *)calloc(HALFSIZE/32, sizeof(u32));
  assert(bits != 0);
  hipMemcpy(bits, ctx.alive.bits, (HALFSIZE/32) * sizeof(u32), hipMemcpyDeviceToHost);
  checkCudaErrors(hipFree(ctx.alive.bits));
  checkCudaErrors(hipFree(ctx.nonleaf.bits));

  u32 cnt = 0;
  for (int i = 0; i < HALFSIZE/32; i++) {
    for (u32 b = ~bits[i]; b; b &= b-1)
      cnt++;
  }
  u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
  printf("final load %d%%\n", load);

  if (load >= 90) {
    printf("overloaded! exiting...");
    exit(0);
  }

  cuckoo_hash &cuckoo = *(new cuckoo_hash());
  node_t us[MAXPATHLEN], vs[MAXPATHLEN];
  for (nonce_t block = 0; block < HALFSIZE; block += 32) {
    for (nonce_t nonce = block; nonce < block+32 && nonce < HALFSIZE; nonce++) {
      if (!(bits[nonce/32] >> (nonce%32) & 1)) {
        node_t u0=sipnode(&ctx.sip_ctx, nonce, 0), v0=sipnode(&ctx.sip_ctx, nonce, 1);
        if (u0 == 0) // ignore vertex 0 so it can be used as nil for cuckoo[]
          continue;
        node_t u = cuckoo[us[0] = u0], v = cuckoo[vs[0] = v0];
        u32 nu = path(cuckoo, u, us), nv = path(cuckoo, v, vs);
        if (us[nu] == vs[nv]) {
          u32 min = nu < nv ? nu : nv;
          for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
          u32 len = nu + nv + 1;
          printf("% 4d-cycle found at %d:%d%%\n", len, 0, (u32)(nonce*100L/HALFSIZE));
          if (len == PROOFSIZE) {
            printf("Solution");
            std::set<edge> cycle;
            u32 n;
            cycle.insert(edge(*us, *vs));
            while (nu--)
              cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
            while (nv--)
              cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
            for (nonce_t nce = n = 0; nce < HALFSIZE; nce++)
              if (!(bits[nce/32] >> (nce%32) & 1)) {
                edge e(sipnode(&ctx.sip_ctx, nce, 0), sipnode(&ctx.sip_ctx, nce, 1));
                if (cycle.find(e) != cycle.end()) {
                  printf(" %lx", nonce);
                  if (PROOFSIZE > 2)
                    cycle.erase(e);
                  n++;
                }
              }
            assert(n==PROOFSIZE);
            printf("\n");
          }
          continue;
        }
        if (nu < nv) {
          while (nu--)
            cuckoo.set(us[nu+1], us[nu]);
          cuckoo.set(u0, v0);
        } else {
          while (nv--)
            cuckoo.set(vs[nv+1], vs[nv]);
          cuckoo.set(v0, u0);
        }
      }
    }
  }
  return 0;
}
