#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"
#include <openssl/sha.h>

// d(evice s)ipnode
#if (__CUDA_ARCH__  >= 320) // redefine ROTL to use funnel shifter, 3% speed gain

static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
static __device__ __forceinline__ void operator^= (uint2 &a, uint2 b) { a.x ^= b.x, a.y ^= b.y; }
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) {
  asm("{\n\tadd.cc.u32 %0,%2,%4;\n\taddc.u32 %1,%3,%5;\n\t}\n\t"
    : "=r"(a.x), "=r"(a.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
}
#undef ROTL
__inline__ __device__ uint2 ROTL(const uint2 a, const int offset) {
  uint2 result;
  if (offset >= 32) {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t" : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}
__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  uint2 nonce = vectorize(2*nce + uorv);
  uint2 v0 = vectorize(keys.k0 ^ 0x736f6d6570736575ULL),
        v1 = vectorize(keys.k1 ^ 0x646f72616e646f6dULL),
        v2 = vectorize(keys.k0 ^ 0x6c7967656e657261ULL),
        v3 = vectorize(keys.k1 ^ 0x7465646279746573ULL) ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= vectorize(0xff);
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return devectorize(v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#else

__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = keys.k0 ^ 0x736f6d6570736575ULL, v1 = keys.k0 ^ 0x646f72616e646f6dULL,
      v2 = keys.k0 ^ 0x6c7967656e657261ULL, v3 = keys.k0 ^ 0x7465646279746573ULL ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#endif

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (NNODES >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// NNODES >> IDXSHIFT == NEDGES >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif

#define NODEBITS (EDGEBITS + 1)
#define NNODES (2 * NEDGES)
#define NODEMASK (NNODES-1)

// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (NODEBITS/3))

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(edge_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(node_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (NEDGES >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
    assert(cuckoo != 0);
  }
  ~cuckoo_hash() {
    free(cuckoo);
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }
  }
};

class cuckoo_ctx {
public:
  siphash_keys sip_keys;
  shrinkingset alive;
  twice_set nonleaf;
  int nthreads;

  cuckoo_ctx(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(char* headernonce, const u32 nonce) {
    ((u32 *)headernonce)[HEADERLEN/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, HEADERLEN, &sip_keys);
  }
};

__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}

__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (edge_t block = id*32; block < NEDGES; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (edge_t nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffs(alive32);
      nonce += ffs; alive32 >>= ffs;
      node_t u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}

u32 path(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      exit(0);
    }
    us[nu++] = u;
  }
  return nu-1;
}

typedef std::pair<node_t,node_t> edge;

#ifndef WIN32
#include <unistd.h>
#else
#include "getopt/getopt.h"
#endif

int main(int argc, char **argv) {
  int nthreads = 16384;
  int trims   = 32;
  int tpb = 0;
  int nonce = 0;
  int range = 1;
  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:n:m:r:t:p:")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        trims = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
      case 'p':
        tpb = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
    }
  }
  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  printf("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with 50%% edges, %d trims, %d threads %d per block\n", trims, nthreads, tpb);

  cuckoo_ctx ctx(nthreads);

  char headernonce[HEADERLEN];
  u32 hdrlen = strlen(header);
  memcpy(headernonce, header, hdrlen);
  memset(headernonce+hdrlen, 0, sizeof(headernonce)-hdrlen);

  u64 edgeBytes = NEDGES/8, nodeBytes = TWICE_WORDS*sizeof(u32);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  printf("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  for (int r = 0; r < range; r++) {
    hipEventRecord(start, NULL);
    checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
    ctx.setheadernonce(headernonce, nonce + r);
    hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);
    for (u32 round=0; round < trims; round++) {
      for (u32 uorv = 0; uorv < 2; uorv++) {
        for (u32 part = 0; part <= PART_MASK; part++) {
          checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
          count_node_deg<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
          kill_leaf_edges<<<nthreads/tpb,tpb >>>(device_ctx, uorv, part);
        }
      }
    }
  
    u64 *bits;
    bits = (u64 *)calloc(NEDGES/64, sizeof(u64));
    assert(bits != 0);
    hipMemcpy(bits, ctx.alive.bits, (NEDGES/64) * sizeof(u64), hipMemcpyDeviceToHost);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    u32 cnt = 0;
    for (int i = 0; i < NEDGES/64; i++)
      cnt += __builtin_popcountll(~bits[i]);
    u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
    printf("nonce %d: %d trims completed in %.3f seconds final load %d%%\n",
            nonce+r, trims, duration / 1000.0f, load);
  
    if (load >= 90) {
      printf("overloaded! exiting...");
      exit(0);
    }
  
    cuckoo_hash &cuckoo = *(new cuckoo_hash());
    node_t us[MAXPATHLEN], vs[MAXPATHLEN];
    for (edge_t block = 0; block < NEDGES; block += 64) {
      u64 alive64 = ~bits[block/64];
      for (edge_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
        u32 ffs = __builtin_ffsll(alive64);
        nonce += ffs; alive64 >>= ffs;
        node_t u0=sipnode(&ctx.sip_keys, nonce, 0), v0=sipnode(&ctx.sip_keys, nonce, 1);
        if (u0) {
          u32 nu = path(cuckoo, u0, us), nv = path(cuckoo, v0, vs);
          if (us[nu] == vs[nv]) {
            u32 min = nu < nv ? nu : nv;
            for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
            u32 len = nu + nv + 1;
            printf("%4d-cycle found at %d:%d%%\n", len, 0, (u32)(nonce*100L/NEDGES));
            if (len == PROOFSIZE) {
              printf("Solution");
              std::set<edge> cycle;
              u32 n = 0;
              cycle.insert(edge(*us, *vs));
              while (nu--)
                cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
              while (nv--)
                cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
              for (edge_t blk = 0; blk < NEDGES; blk += 64) {
                u64 alv64 = ~bits[blk/64];
                for (edge_t nce = blk-1; alv64; ) { // -1 compensates for 1-based ffs
                  u32 ffs = __builtin_ffsll(alv64);
                  nce += ffs; alv64 >>= ffs;
                  edge e(sipnode(&ctx.sip_keys, nce, 0), sipnode(&ctx.sip_keys, nce, 1));
                  if (cycle.find(e) != cycle.end()) {
                    printf(" %jx", (uintmax_t)nce);
                    if (PROOFSIZE > 2)
                      cycle.erase(e);
                    n++;
                  }
                  if (ffs & 64) break; // can't shift by 64
                }
              }
              assert(n==PROOFSIZE);
              printf("\n");
            }
          } else if (nu < nv) {
            while (nu--)
              cuckoo.set(us[nu+1], us[nu]);
            cuckoo.set(u0, v0);
          } else {
            while (nv--)
              cuckoo.set(vs[nv+1], vs[nv]);
            cuckoo.set(v0, u0);
          }
        }
        if (ffs & 64) break; // can't shift by 64
      }
    }
  }
  checkCudaErrors(hipFree(ctx.alive.bits));
  checkCudaErrors(hipFree(ctx.nonleaf.bits));
  return 0;
}
